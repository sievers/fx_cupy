//nvcc -o libfx_cupy.so fx_cupy.cu -shared -Xcompiler -fPIC -lgomp

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <omp.h>
#include <hip/hip_complex.h>


#define KMAX 4

extern "C"
{
void h2d(void *hptr, void *dptr, int nbyte)
{
  if (hipMemcpy(dptr,hptr,(size_t)nbyte,hipMemcpyHostToDevice)!=hipSuccess)
    fprintf(stderr,"Failed copy on h2d with %d bytes.\n",nbyte);
}
}

/*--------------------------------------------------------------------------------*/
template<typename T>
__global__
void conv_cols_gpu(T *in, float *out, float *win, int n, int m, int kk)
//__global__
//void conv_cols_gpu(int8_t *in, float *out, float *win, int n, int m, int kk)

{
  float mywin[KMAX];
  float mydat[KMAX];
  int di=blockDim.x*gridDim.x;
  for (int i=blockDim.x*blockIdx.x+threadIdx.x;i<n;i+=di)
    {
      //copy window
      for (int j=0;j<kk;j++)
	mywin[j]=win[i+j*n];
      //copy first bit of data
      for (int j=0;j<kk-1;j++)
	mydat[j]=in[i+j*n];

      //loop over rows
      for (int j=kk-1;j<m;j++) {
	mydat[kk-1]=in[i+(j)*n];
	float tmp=0;
	for (int k=0;k<kk;k++) 
	  tmp+=mywin[k]*mydat[k];
	//out[i+(j+1-kk)*n]=mydat[0];
	out[i+(j+1-kk)*n]=tmp;
	//out[i+(j+1-kk)*n]=mywin[1];
	for (int k=0;k<kk-1;k++)
	  mydat[k]=mydat[k+1];

	
      }
      
    }
}
/*--------------------------------------------------------------------------------*/
/*--------------------------------------------------------------------------------*/
template<typename T>
__global__
void conv_cols_complex_gpu(T *in, hipFloatComplex *out, float *win, int n, int m, int kk)
{
  hipFloatComplex mywin[KMAX];
  hipFloatComplex mydat[KMAX];
  int di=blockDim.x*gridDim.x;
  for (int i=blockDim.x*blockIdx.x+threadIdx.x;i<n;i+=di)
    {
      //copy window
      for (int j=0;j<kk;j++)
	mywin[j]=make_hipFloatComplex(win[i+j*n],0);
      //copy first bit of data
      for (int j=0;j<kk-1;j++)  {
	T tmp;
	tmp=in[i+j*n];
	//mydat[j]=toComplex(in[i+j*n]);
	mydat[j]=make_hipFloatComplex(tmp.x,tmp.y);
      }

      //loop over rows
      for (int j=kk-1;j<m;j++) {
	{
	  T tmp;
	  tmp=in[i+(j)*n];
	  mydat[kk-1]=make_hipFloatComplex(tmp.x,tmp.y);
	  //mydat[kk-1]=in[i+(j)*n];
	}
	hipFloatComplex tmp=make_hipFloatComplex(0,0);
	for (int k=0;k<kk;k++) 
	  tmp=hipCaddf(tmp,hipCmulf(mywin[k],mydat[k]));
	out[i+(j+1-kk)*n]=tmp;
	for (int k=0;k<kk-1;k++)
	  mydat[k]=mydat[k+1];
	
      }
      
    }
}
/*--------------------------------------------------------------------------------*/
extern "C"
{
void conv_cols(void *in, float *out, float *win, int n, int m, int kk, int elemsize)
{
  //printf("elemsize is %d\n",elemsize);
  switch(elemsize) {
  case 1:
    conv_cols_gpu<<<128,256>>>((int8_t *)in,out,win,n,m,kk);
    break;
  case 2:
    conv_cols_gpu<<<256,256>>>((int16_t *)in,out,win,n,m,kk);
    break;
  case 4:
    conv_cols_gpu<<<256,256>>>((int *)in,out,win,n,m,kk);
    break;
  case -4:
    conv_cols_gpu<<<256,256>>>((float *)in,out,win,n,m,kk);
    break;
  default:
    fprintf(stderr,"Unhandled element size %d in conv_cols_gpu.\n",elemsize);
    break;
  }
}
}
/*--------------------------------------------------------------------------------*/
extern "C"
{
void conv_cols_complex(void *in, hipFloatComplex *out, float *win, int n, int m, int kk, int elemsize)
{
  //printf("elemsize is %d\n",elemsize);
  switch(elemsize) {
  case -8:
    conv_cols_complex_gpu<<<128,256>>>((hipFloatComplex *)in,out,win,n,m,kk);
    break;
  case -16:
    conv_cols_complex_gpu<<<256,256>>>((hipDoubleComplex *)in,out,win,n,m,kk);
    break;
  default:
    fprintf(stderr,"Unhandled element size %d in conv_cols_complex_gpu.\n",elemsize);
    break;
  }
}
}
/*--------------------------------------------------------------------------------*/

#define BS_CHERK 8

__global__
void cherk_batched_gpu(hipFloatComplex *in, hipFloatComplex *out, int nmat, int n, int k)
{
  int nblock=k/BS_CHERK;
  if ((nblock*BS_CHERK)<n)
    nblock++;
  for (int imat=blockIdx.x;imat<nmat;imat+=gridDim.x)
    {
      __shared__ hipFloatComplex patch[BS_CHERK][BS_CHERK];

      hipFloatComplex tmp=make_hipFloatComplex(0,0);
      for (int i=0;i<nblock;i++) {
	int myind=i*BS_CHERK+threadIdx.x;
	if (myind<k)
	  patch[threadIdx.y][threadIdx.x]=in[threadIdx.y*k+myind+imat*n*k];
	else
	  patch[threadIdx.y][threadIdx.x]=make_hipFloatComplex(0,0);
	__syncthreads();
	for (int j=0;j<BS_CHERK;j++)
	  tmp=hipCaddf(tmp,hipCmulf(patch[threadIdx.x][j],hipConjf(patch[threadIdx.y][j])));
	__syncthreads();
	//tmp=patch[0][0];
	//__syncthreads();
      }
      out[imat*n*n+threadIdx.x*BS_CHERK+threadIdx.y]=tmp;
      //out[imat*n*n+threadIdx.x*BS_CHERK+threadIdx.y]=make_cuFloatComplex(1,0);
      __syncthreads();
      //out[0]=make_cuFloatComplex(1,0);
      
    }
}
/*--------------------------------------------------------------------------------*/
__global__
void cherk_batched_gpu_transpose(hipFloatComplex *in, hipFloatComplex *out, int nmat, int n, int k)
///do a batched cherk, but expect data ordering as per PFB, so (n,k,nmat)
{
  int nblock=k/BS_CHERK;
  if ((nblock*BS_CHERK)<n)
    nblock++;
  for (int imat=blockIdx.x;imat<nmat;imat+=gridDim.x)
    {
      __shared__ hipFloatComplex patch[BS_CHERK][BS_CHERK];

      hipFloatComplex tmp=make_hipFloatComplex(0,0);
      for (int i=0;i<nblock;i++) {
	
	int myind=i*BS_CHERK+threadIdx.x;
	if (myind<k)
	  patch[threadIdx.y][threadIdx.x]=in[threadIdx.y*k+myind+imat*n*k];
	else
	  patch[threadIdx.y][threadIdx.x]=make_hipFloatComplex(0,0);

	__syncthreads();
	for (int j=0;j<BS_CHERK;j++)
	  tmp=hipCaddf(tmp,hipCmulf(patch[threadIdx.x][j],hipConjf(patch[threadIdx.y][j])));
	__syncthreads();
	//tmp=patch[0][0];
	//__syncthreads();
      }
      out[imat*n*n+threadIdx.x*BS_CHERK+threadIdx.y]=tmp;
      //out[imat*n*n+threadIdx.x*BS_CHERK+threadIdx.y]=make_cuFloatComplex(1,0);
      __syncthreads();
      //out[0]=make_cuFloatComplex(1,0);
      
    }
}
/*--------------------------------------------------------------------------------*/

extern "C"
{
void cherk_batched(hipFloatComplex *in, hipFloatComplex *out, int nmat, int n, int k)
{
  //cherk_batched_gpu<<<512,(BS_CHERK,BS_CHERK)>>>(in,out,nmat,n,k);
  dim3 threadsize;
  threadsize.x=BS_CHERK;
  threadsize.y=BS_CHERK;
  threadsize.z=1;
  cherk_batched_gpu<<<256,threadsize>>>(in,out,nmat,n,k);
}
}
/*--------------------------------------------------------------------------------*/
__global__
void apply_pfb_win(int8_t *din, float *dout, float *win, int ntap, int nchan, int npol, int nblock, int nn)
{
  
}

